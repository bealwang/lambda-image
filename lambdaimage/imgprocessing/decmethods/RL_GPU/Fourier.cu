#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
const int THREADS=64;
__global__ void kernel_divideStable2(float *data1, float *data2, float threadshold, int len);
__global__ void kernel_multiply_complex(float *data1, float *data2, int len);
__global__ void kernel_multiply_real(float *data1, float *data2, int len);
__global__ void kernel_multiplyConj(float *data1, float *data2, int len);
__global__ void kernel_copy(float *data1, float *data2, int len);
using namespace std;
void inv_forwardDFT(float *data, int nx, int ny, int nz){
    hipfftReal *a=(hipfftReal*)data;
    hipfftComplex *A=(hipfftComplex*)data;
    hipfftHandle plan_forward;
    
    if(nz==1){
        hipfftPlan2d(&plan_forward, nx, ny, HIPFFT_R2C);
        hipfftExecR2C(plan_forward, a, A);
    }
    else{
//        cout<<"nx= "<<nx<<" ny= "<<ny<<" nz= "<<nz<<endl;
        hipfftPlan3d(&plan_forward, nx, ny, nz, HIPFFT_R2C);
        hipfftExecR2C(plan_forward, a, A);
    }
    hipfftDestroy(plan_forward);
}

void inv_inverseDFT(float *data, int nx, int ny, int nz){
    hipfftReal *a=(hipfftReal*)data;
    hipfftComplex *A=(hipfftComplex*)data;
    hipfftHandle plan_backward;
    
    if(nz==1){
        hipfftPlan2d(&plan_backward, nx, ny, HIPFFT_C2R);
        hipfftExecC2R(plan_backward, A, a);
    }
    else{
        hipfftPlan3d(&plan_backward, nx, ny, nz, HIPFFT_C2R);
        hipfftExecC2R(plan_backward, A, a);
    }
    hipfftDestroy(plan_backward);
}

void inv_divideStable2(float *data1, float *data2, float threadshold, int len){
    unsigned long int block=(len+THREADS-1)/THREADS;
    kernel_divideStable2<<<block, THREADS>>>(data1, data2, threadshold, len);
} 
__global__ void kernel_divideStable2(float *data1, float *data2, float threadshold, int len){
    int offset=blockDim.x*blockIdx.x+threadIdx.x;
    if(offset>=len) return;

    if(data2[offset]<threadshold)
        data2[offset]=(float)0;
    else
        data2[offset]=data1[offset]/data2[offset];
}

void inv_multiply_complex(float * data1, float *data2, int len){
    unsigned long int block=(len/2+THREADS-1)/THREADS;
    kernel_multiply_complex<<<block, THREADS>>>(data1, data2, len/2);

}
__global__ void kernel_multiply_complex(float *data1, float *data2, int len){
        int offset=blockDim.x*blockIdx.x+threadIdx.x;
        if(offset>=len) return;

        float temp1=data1[(2*offset)];
        float temp2=data2[(2*offset)];
        data1[(2*offset)]=(temp1*temp2-data1[(2*offset+1)]*data2[(2*offset+1)]);
        data1[(2*offset+1)]=(temp1*data2[(2*offset+1)]+data1[(2*offset+1)]*temp2);
}

void inv_multiply_real(float *data1, float *data2, int len){
    unsigned long int block=(len+THREADS-1)/THREADS;
    kernel_multiply_real<<<block, THREADS>>>(data1, data2, len);
}
__global__ void kernel_multiply_real(float *data1, float *data2, int len){
    int offset=blockDim.x*blockIdx.x+threadIdx.x;
    if(offset>=len) return;
    data1[offset] *= data2[offset];
}
    
void inv_multiplyConj(float *data1, float *data2, int len){
    unsigned long int block=(len/2+THREADS-1)/THREADS;
    kernel_multiplyConj<<<block, THREADS>>>(data1, data2, len/2);
}
__global__ void kernel_multiplyConj(float *data1, float *data2, int len){
    int offset=blockDim.x*blockIdx.x+threadIdx.x;
    if(offset>=len) return;

    float temp1 = data1[(2 * offset)];
    float temp2 = data2[(2 * offset)];
    data1[(2*offset)] = (temp1*temp2+data1[(2*offset+1)]*data2[(2*offset+1)]);
    data1[(2*offset+1)] = (-temp1*data2[(2*offset+1)]+data1[(2*offset+1)]*temp2);
}

void inv_copy(float *data1, float *data2, int len){
    unsigned long int block=(len+THREADS-1)/THREADS;
    kernel_copy<<<block, THREADS>>>(data1, data2, len);
}
__global__ void kernel_copy(float *data1, float *data2, int len){
    int offset=blockDim.x*blockIdx.x+threadIdx.x;
    if(offset>=len) return;
    data1[offset]=data2[offset];
}
